#include "hip/hip_runtime.h"
#include "optkit_dense.h"
#include "optkit_defs_gpu.h"
#include "optkit_thrust.hpp"


#ifdef __cplusplus
extern "C" {
#endif

void denselib_version(int * maj, int * min, int * change, int * status)
{
        * maj = OPTKIT_VERSION_MAJOR;
        * min = OPTKIT_VERSION_MINOR;
        * change = OPTKIT_VERSION_CHANGE;
        * status = (int) OPTKIT_VERSION_STATUS;
}


__global__ void _get_cuda_nan(ok_float * val)
{
        *val = OK_CUDA_NAN;
}

inline ok_float get_cuda_nan()
{
        ok_float res;
        ok_float * res_dev;

        ok_alloc_gpu(res_dev, 1 * sizeof(ok_float));
        hipMemcpy(&res, res_dev, 1 * sizeof(ok_float), hipMemcpyDeviceToHost);
        ok_free_gpu(res_dev);

        return res;
}

/*
 * VECTOR CUDA helper methods
 * ==========================
 */
__global__ void __vector_set(ok_float * data, ok_float val, size_t stride,
        size_t size)
{
        uint i, thread_id = blockIdx.x * blockDim.x + threadIdx.x;
        for (i = thread_id; i < size; i += gridDim.x * blockDim.x)
        data[i * stride] = val;
}

void __vector_set_all(vector * v, ok_float x)
{
        uint grid_dim = calc_grid_dim(v->size);
        __vector_set<<<grid_dim, kBlockSize>>>(v->data, x, v->stride, v->size);
}

__global__ void __strided_memcpy(ok_float * x, size_t stride_x,
        const ok_float * y, size_t stride_y, size_t size)
{
        uint i, tid = blockIdx.x * blockDim.x + threadIdx.x;
        for (i = tid; i < size; i += gridDim.x * blockDim.x)
        x[i * stride_x] = y[i * stride_y];
}


/*
 * VECTOR methods
 * ==============
 */
inline int __vector_exists(vector * v)
{
        if (v == OK_NULL) {
                printf("Error: cannot write to uninitialized vector pointer\n");
                return 0;
        } else {
                return 1;
        }
}

void vector_alloc(vector * v, size_t n)
{
        if (!__vector_exists(v))
                return;
        v->size=n;
        v->stride=1;
        ok_alloc_gpu(v->data, n * sizeof(ok_float));
}

void vector_calloc(vector * v, size_t n)
{
        vector_alloc(v, n);
        __vector_set_all(v, ok_float(0));
}

void vector_free(vector * v)
{
        if (v != OK_NULL)
                if (v->data != OK_NULL) ok_free_gpu(v->data);
        v->size = (size_t) 0;
        v->stride = (size_t) 0;
}

void vector_set_all(vector * v, ok_float x)
{
        __vector_set_all(v, x);
}

void vector_subvector(vector * v_out, vector * v_in, size_t offset, size_t n)
{
        if (!__vector_exists(v_out))
                return;
        v_out->size=n;
        v_out->stride=v_in->stride;
        v_out->data=v_in->data + offset * v_in->stride;
}

vector vector_subvector_gen(vector * v_in, size_t offset, size_t n)
{
        return (vector){
                .size = n,
                .stride = v_in->stride,
                .data = v_in->data + offset * v_in->stride
        };
}

void vector_view_array(vector * v, ok_float * base, size_t n)
{
          if (!__vector_exists(v))
                return;
          v->size=n;
          v->stride=1;
          v->data=base;
}


void vector_memcpy_vv(vector * v1, const vector * v2)
{
        uint grid_dim;
        if ( v1->stride == 1 && v2->stride == 1) {
                ok_memcpy_gpu(v1->data, v2->data, v1->size * sizeof(ok_float));
        } else {
                grid_dim = calc_grid_dim(v1->size);
                __strided_memcpy<<<grid_dim, kBlockSize>>>(v1->data, v1->stride,
                        v2->data, v2->stride, v1->size);
        }
}

void vector_memcpy_va(vector * v, const ok_float *y, size_t stride_y)
{
        uint i;
        if (v->stride == 1 && stride_y == 1)
                ok_memcpy_gpu(v->data, y, v->size * sizeof(ok_float));
        else
                for (i = 0; i < v->size; ++i)
        ok_memcpy_gpu(v->data + i * v->stride, y + i * stride_y,
        sizeof(ok_float));
}

void vector_memcpy_av(ok_float *x, const vector *v, size_t stride_x)
{
        uint i;
        if (v->stride == 1 && stride_x == 1)
                ok_memcpy_gpu(x, v->data, v->size * sizeof(ok_float));
        else
                for (i = 0; i < v->size; ++i)
        ok_memcpy_gpu(x + i * stride_x, v->data + i * v->stride,
        sizeof(ok_float));
}

void vector_print(const vector * v)
{
        uint i;
        ok_float v_host[v->size];
        vector_memcpy_av(v_host, v, 1);
        for (i = 0; i < v->size; ++i)
                printf("%e ", v_host[i]);
        printf("\n");
}

void vector_scale(vector * v, ok_float x)
{
        __thrust_vector_scale(v, x);
        CUDA_CHECK_ERR;
}

void vector_add(vector * v1, const vector * v2)
{
        __thrust_vector_add(v1, v2);
        CUDA_CHECK_ERR;
}

void vector_sub(vector * v1, const vector * v2)
{
        __thrust_vector_sub(v1, v2);
        CUDA_CHECK_ERR;
}

void vector_mul(vector * v1, const vector * v2)
{
        __thrust_vector_mul(v1, v2);
        CUDA_CHECK_ERR;
}

void vector_div(vector * v1, const vector * v2)
{
        __thrust_vector_div(v1, v2);
        CUDA_CHECK_ERR;
}

void vector_add_constant(vector * v, const ok_float x)
{
        __thrust_vector_add_constant(v, x);
        CUDA_CHECK_ERR;
        }

void vector_abs(vector * v)
{
        __thrust_vector_abs(v);
        CUDA_CHECK_ERR;
}

void vector_recip(vector * v)
{
        __thrust_vector_recip(v);
        CUDA_CHECK_ERR;
}

void vector_sqrt(vector * v)
{
        __thrust_vector_sqrt(v);
        CUDA_CHECK_ERR;
}

void vector_pow(vector * v, const ok_float x)
{
        __thrust_vector_pow(v, x);
        CUDA_CHECK_ERR;
}


/*
 * MATRIX CUDA helper methods
 * ==========================
 */

/* row major setter */
__global__ void __matrix_set_r(ok_float * data, ok_float x, size_t stride,
        size_t size1, size_t size2)
{
        uint i, j;
        uint thread_id_row = blockIdx.x * blockDim.x + threadIdx.x;
        uint thread_id_col = blockIdx.y * blockDim.y + threadIdx.y;
        uint incr_x = gridDim.x * blockDim.x;
        uint incr_y = gridDim.y * blockDim.y;
        for (i = thread_id_row; i < size1; i += incr_x)
                for (j = thread_id_col; j < size2; j += incr_y)
                        data[i * stride + j] = x;
}

/* column major setter */
__global__ void __matrix_set_c(ok_float * data, ok_float x, size_t stride,
        size_t size1, size_t size2)
{
        uint i, j;
        uint thread_id_row = blockIdx.x * blockDim.x + threadIdx.x;
        uint thread_id_col = blockIdx.y * blockDim.y + threadIdx.y;
        uint incr_x = gridDim.x * blockDim.x;
        uint incr_y = gridDim.y * blockDim.y;
        for (j = thread_id_col; j < size2; j += incr_y)
                for (i = thread_id_row; i < size1; i += incr_x)
                        data[i + j * stride] = x;
}


void __matrix_set_all(matrix * A, ok_float x)
{
        uint grid_dimx = calc_grid_dim(A->size1);
        uint grid_dimy = calc_grid_dim(A->size2);
        dim3 grid_dim(grid_dimx, grid_dimy, 1u);
        dim3 block_dim(kBlockSize, kBlockSize - 1, 1u);

        if (A->order == CblasRowMajor)
                __matrix_set_r<<<grid_dim, block_dim>>>(A->data, x, A->ld,
                        A->size1, A->size2);
        else
                __matrix_set_c<<<grid_dim, block_dim>>>(A->data, x, A->ld,
                        A->size1, A->size2);
        CUDA_CHECK_ERR;
}

__global__ void __matrix_add_constant_diag(ok_float * data, ok_float x,
        size_t stride)
{
        uint i = blockIdx.x * blockDim.x + threadIdx.x;
        data[i * stride + i] += x;
}


/* row major data retrieval */
__device__ inline ok_float& __matrix_get_r(ok_float * A, uint i, uint j,
        uint stride)
{
        return A[i * stride + j];
}


/* column major data retrieval */
__device__ inline ok_float& __matrix_get_c(ok_float * A, uint i, uint j,
        uint stride)
{
        return A[i + j * stride];
}


/*
 * MATRIX methods
 * ==============
 */
inline int __matrix_exists(matrix * A)
{
        if (A == OK_NULL) {
                printf("Error: cannot write to uninitialized matrix pointer\n");
                return 0;
        } else {
                return 1;
        }
}

void matrix_alloc(matrix * A, size_t m, size_t n, enum CBLAS_ORDER ord)
{
        A->size1 = m;
        A->size2 = n;
        ok_alloc_gpu(A->data, m * n * sizeof(ok_float));
        A->ld = (ord == CblasRowMajor) ? n : m;
        A->order = ord;
}

void matrix_calloc(matrix * A, size_t m, size_t n, enum CBLAS_ORDER ord)
{
        if (!__matrix_exists(A))
                return;
        matrix_alloc(A, m, n, ord);
        hipMemset(A->data, 0, m * n * sizeof(ok_float));
        CUDA_CHECK_ERR;
}

void matrix_free(matrix * A)
{
        if (A == OK_NULL || A->data != OK_NULL)
                return;
        ok_free_gpu(A->data);
        A->size1 = (size_t) 0;
        A->size2 = (size_t) 0;
        A->ld = (size_t) 0;

}

void matrix_submatrix(matrix * A_sub, matrix * A, size_t i, size_t j, size_t n1,
        size_t n2)
{
        __matrix_exists(A_sub);
        A_sub->size1 = n1;
        A_sub->size2 = n2;
        A_sub->ld = A->ld;
        A_sub->data = (A->order == CblasRowMajor) ?
                      A->data + (i * A->ld) + j : A->data + i + (j * A->ld);
        A_sub->order = A->order;
}

matrix matrix_submatrix_gen(matrix * A, size_t i, size_t j, size_t n1,
        size_t n2)
{
        return (matrix){
                .size1 = n1,
                .size2 = n2,
                .ld = A->ld,
                .data = (A->order == CblasRowMajor) ?
                        A->data + (i * A->ld) + j : A->data + i + (j * A->ld),
                .order = A->order
        };
}

void matrix_row(vector * row, matrix * A, size_t i)
{
        if (!__vector_exists(row))
                return;
        row->size = A->size2;
        row->stride = (A->order == CblasRowMajor) ? 1 : A->ld;
        row->data = (A->order == CblasRowMajor) ?
                    A->data + (i * A->ld) : A->data + i;
}

void matrix_column(vector * col, matrix *A, size_t j)
{
        if (!__vector_exists(col))
                return;
        col->size = A->size1;
        col->stride = (A->order == CblasRowMajor) ? A->ld : 1;
        col->data = (A->order == CblasRowMajor) ?
                    A->data + j : A->data + (j * A->ld);
}

void matrix_diagonal(vector * diag, matrix *A)
{
        if (!__vector_exists(diag))
                return;
        diag->data = A->data;
        diag->stride = A->ld + 1;
        diag->size = (size_t) (A->size1 <= A->size2) ? A->size1 : A->size2;
}

void matrix_cast_vector(vector * v, matrix * A)
{
        v->size = A->size1 * A->size2;
        v->stride = 1;
        v->data = A->data;
}

void matrix_view_array(matrix * A, const ok_float *base, size_t n1,
                       size_t n2, enum CBLAS_ORDER ord)
{
        if (!__matrix_exists(A))
                return;
        A->size1 = n1;
        A->size2 = n2;
        A->data = (ok_float *) base;
        A->ld = (ord == CblasRowMajor) ? n2 : n1;
        A->order = ord;
}

void matrix_set_all(matrix * A, ok_float x)
{
        __matrix_set_all(A, x);
}


void matrix_memcpy_mm(matrix * A, const matrix * B)
{
        uint i, j, grid_dim;
        if (A->size1 != B->size1) {
                printf("error: m-dimensions must match for matrix memcpy\n");
                return;
        } else if (A->size2 != B->size2) {
                printf("error: n-dimensions must match for matrix memcpy\n");
                return;
        }

        if (A->order == B->order) {
                ok_memcpy_gpu(A->data, B->data,
                              A->size1 * A->size2 * sizeof(ok_float));
        } else if (A->order == CblasRowMajor) {
                /* A row major, B column major */
                grid_dim = calc_grid_dim(A->size1);
                for (i = 0; i < A->size1; ++i)
                        __strided_memcpy<<<grid_dim, kBlockSize>>>(
                                A->data + i * A->size2, 1,
                                B->data + i, A->ld, A->size2);
        } else {
                /* A column major, B row major */
                grid_dim = calc_grid_dim(A->size2);
                for (j= 0; j < A->size2; ++j)
                        __strided_memcpy<<<grid_dim, kBlockSize>>>(
                                A->data + j * A->size1, 1,
                                B->data + j, A->ld, A->size1);
        }
        CUDA_CHECK_ERR;
}

/*
 * Handle the following cases:
 *      A->order = ord
 *      A->order != ord, ord == CblasColMajor (A row major, B col major)
 *      A->order != ord, ord == CblasRowMajor (A col major, B row major)
 */
void matrix_memcpy_ma(matrix * A, const ok_float * B,
	const enum CBLAS_ORDER ord)
{
        uint i, j, grid_dim;
        ok_float * row, * col;

        if (A->order == ord) {
        	if (ord == CblasRowMajor)
	        	for (i = 0; i < A->size1; ++i)
		                ok_memcpy_gpu(A->data + i * A->ld,
		                	B + i * A->size2,
		                	A->size2 * sizeof(ok_float));
		else
	        	for (j = 0; j < A->size2; ++j)
		                ok_memcpy_gpu(A->data + j * A->ld,
		                	B + j * A->size1,
		                	A->size1 * sizeof(ok_float));
        } else if (ord == CblasColMajor) {
                ok_alloc_gpu(col, A->size1 * sizeof(ok_float));
                grid_dim = calc_grid_dim(A->size1);
                for (j = 0; j < A->size2; ++j) {
                        ok_memcpy_gpu(col, B + j * A->size1,
                                      A->size1 * sizeof(ok_float));
                        __strided_memcpy<<<grid_dim, kBlockSize>>>(A->data + j,
                                A->ld, col, 1, A->size1);
                }
                ok_free_gpu(col);
        } else {
                ok_alloc_gpu(row, A->size2 * sizeof(ok_float));
                grid_dim = calc_grid_dim(A->size2);
                for (i = 0; i < A->size1; ++i) {
                        ok_memcpy_gpu(row, B + i * A->size2,
                                      A->size2 * sizeof(ok_float));
                        __strided_memcpy<<<grid_dim, kBlockSize>>>(A->data + i,
                                A->ld, row, 1, A->size2);
                }
                ok_free_gpu(row);
        }
        CUDA_CHECK_ERR;
}

/*
 * Handle the following cases:
 *      ord = B->order
 *      ord != B->order, ord == CblasRowMajor (A row major, B col major)
 *      ord != B->ord, order == CblasColMajor (A col major, B row major)
 */
void matrix_memcpy_am(ok_float * A, const matrix * B,
	const enum CBLAS_ORDER ord)
{
        uint i, j, grid_dim;
        ok_float * row, * col;
        if (ord == B->order) {
        	if (ord == CblasRowMajor)
	        	for (i = 0; i < B->size1; ++i)
		                ok_memcpy_gpu(A + i * B->size2,
		                	B->data + i * B->ld,
		                	B->size2 * sizeof(ok_float));
		else
	        	for (j = 0; j < B->size2; ++j)
		                ok_memcpy_gpu(A + j * B->size1,
		                	B->data + j * B->ld,
		                	B->size1 * sizeof(ok_float));
        } else if (ord == CblasRowMajor) {
                ok_alloc_gpu(row, B->size2 * sizeof(ok_float));
                grid_dim = calc_grid_dim(B->size2);
                for (i = 0; i < B->size1; ++i) {
                        __strided_memcpy<<<grid_dim, kBlockSize>>>(row, 1,
                                B->data + i, B->ld, B->size2);
                        ok_memcpy_gpu(A + i * B->size2, row,
                                      B->size2 * sizeof(ok_float));
                }
                ok_free_gpu(row);
        } else {
                ok_alloc_gpu(col, B->size1 * sizeof(ok_float));
                grid_dim = calc_grid_dim(B->size1);
                for (j = 0; j < B->size2; ++j) {
                        __strided_memcpy<<<grid_dim, kBlockSize>>>(col, 1,
                                B->data + j, B->ld, B->size1);
                        ok_memcpy_gpu(A + j * B->size1, col,
                                      B->size1 * sizeof(ok_float));
                }
                ok_free_gpu(col);
        }
        CUDA_CHECK_ERR;
}

void matrix_print(matrix * A)
{
        ok_float row_host[A->size2];
        vector row = (vector){0, 0, OK_NULL};

        for (uint i = 0; i < A->size1; ++i) {
        	matrix_row(&row, A, i);
        	vector_memcpy_av(row_host, &row, 1);
                for (uint j = 0; j < A->size2; ++j)
                        printf("%0.2e ", row_host[j]);
                printf("\n");
        }
        printf("\n");
}

void matrix_scale(matrix * A, ok_float x)
{
        size_t i;
        vector row_col = (vector){0, 0, OK_NULL};
        if (A->order == CblasRowMajor)
                for(i = 0; i < A->size1; ++i) {
                        matrix_row(&row_col, A, i);
                        vector_scale(&row_col, x);
                }
        else
                for(i = 0; i < A->size2; ++i) {
                        matrix_column(&row_col, A, i);
                        vector_scale(&row_col, x);
                }
}

void matrix_scale_left(matrix * A, const vector * v)
{
        size_t i;
        vector col = (vector){0, 0, OK_NULL};
        for(i = 0; i < A->size2; ++i) {
                matrix_column(&col, A, i);
                vector_mul(&col, v);
        }
}

void matrix_scale_right(matrix * A, const vector * v)
{
        size_t i;
        vector row = (vector){0, 0, OK_NULL};
        for(i = 0; i < A->size1; ++i) {
                matrix_row(&row, A, i);
                vector_mul(&row, v);
        }
}

void matrix_abs(matrix * A)
{
        size_t i;
        vector row_col = (vector){0,0,OK_NULL};
        if (A->order == CblasRowMajor)
                for(i = 0; i < A->size1; ++i) {
                        matrix_row(&row_col, A, i);
                        vector_abs(&row_col);
                }
        else
                for(i = 0; i < A->size2; ++i) {
                        matrix_column(&row_col, A, i);
                        vector_abs(&row_col);
                }
}

void matrix_pow(matrix * A, const ok_float x)
{
        size_t i;
        vector row_col = (vector){0,0,OK_NULL};
        if (A->order == CblasRowMajor)
                for(i = 0; i < A->size1; ++i) {
                        matrix_row(&row_col, A, i);
                        vector_pow(&row_col, x);
                }
        else
                for(i = 0; i < A->size2; ++i) {
                        matrix_column(&row_col, A, i);
                        vector_pow(&row_col, x);
                }
}

int __matrix_order_compat(const matrix * A, const matrix * B, const char * nm_A,
        const char * nm_B, const char * nm_routine)
{
        if (A->order == B->order)
            return 1;

        printf("OPTKIT ERROR (%s) matrices %s and %s must have same layout.\n",
                 nm_routine, nm_A, nm_B);
        return 0;
}

/*
 * BLAS routines
 * =============
 */
ok_status blas_make_handle(void ** handle)
{
        hipblasStatus_t status;
        hipblasHandle_t * hdl;
        hdl = (hipblasHandle_t *) malloc(sizeof(hipblasHandle_t));
        status = hipblasCreate(hdl);
        if (status != HIPBLAS_STATUS_SUCCESS) {
                printf("CUBLAS initialization failed\n");
                ok_free(hdl);
                *handle = OK_NULL;
                return OPTKIT_ERROR_CUBLAS;
        } else {
                *handle = (void *) hdl;
                return OPTKIT_SUCCESS;
        }
}

ok_status blas_destroy_handle(void * handle)
{
        hipblasDestroy(*(hipblasHandle_t *) handle);
        CUDA_CHECK_ERR;
        ok_free(handle);
        return OPTKIT_SUCCESS;
}


/* BLAS LEVEL 1 */
void blas_axpy(void * linalg_handle, ok_float alpha, const vector *x, vector *y)
{
        if (!linalg_handle)
                return;
        CUBLAS(axpy)(*(hipblasHandle_t *) linalg_handle, (int) x->size, &alpha,
                x->data, (int) x->stride, y->data, (int) y->stride);
        CUDA_CHECK_ERR;
}

ok_float blas_nrm2(void * linalg_handle, const vector *x)
{
        ok_float result = kZero;
        if (!linalg_handle)
                return get_cuda_nan();
        CUBLAS(nrm2)(*(hipblasHandle_t *) linalg_handle, (int) x->size, x->data,
                (int) x->stride, &result);
        CUDA_CHECK_ERR;
        return result;
}

void blas_scal(void * linalg_handle, const ok_float alpha, vector *x)
{
        if (!linalg_handle)
                return;
        CUBLAS(scal)(*(hipblasHandle_t *) linalg_handle, (int) x->size, &alpha,
                x->data, (int) x->stride);
        CUDA_CHECK_ERR;
}

ok_float blas_asum(void * linalg_handle, const vector * x)
{
        ok_float result = kZero;
        if (!linalg_handle)
                return get_cuda_nan();
        CUBLAS(asum)(*(hipblasHandle_t *) linalg_handle, (int) x->size, x->data,
                (int) x->stride, &result);
        CUDA_CHECK_ERR;
        return result;
}

ok_float blas_dot(void * linalg_handle, const vector * x, const vector * y)
{
        ok_float result = kZero;
        if (!linalg_handle)
                return get_cuda_nan();
        CUBLAS(dot)(*(hipblasHandle_t *) linalg_handle, (int) x->size, x->data,
                (int) x->stride, y->data, (int) y->stride, &result);
        CUDA_CHECK_ERR;
        return result;
}

void blas_dot_inplace(void * linalg_handle, const vector * x, const vector * y,
        ok_float * deviceptr_result)
{
        CUBLAS(dot)(*(hipblasHandle_t *) linalg_handle, (int) x->size, x->data,
                (int) x->stride, y->data, (int) y->stride, deviceptr_result);
        CUDA_CHECK_ERR;
}

/* BLAS LEVEL 2 */

void blas_gemv(void * linalg_handle, enum CBLAS_TRANSPOSE transA,
	ok_float alpha, const matrix *A, const vector *x, ok_float beta,
	vector *y)
{
        hipblasOperation_t tA;
        int s1, s2;

        if (A->order == CblasColMajor)
                tA = (transA == CblasTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
        else
                tA = (transA == CblasTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;

        s1 = (A->order == CblasRowMajor) ? (int) A->size2 : (int) A->size1;
        s2 = (A->order == CblasRowMajor) ? (int) A->size1 : (int) A->size2;

        if (!linalg_handle)
                return;

        CUBLAS(gemv)(*(hipblasHandle_t *) linalg_handle, tA, s1, s2, &alpha,
                A->data, (int) A->ld, x->data, (int) x->stride, &beta, y->data,
                (int) y->stride);
        CUDA_CHECK_ERR;
}

void blas_trsv(void * linalg_handle, enum CBLAS_UPLO uplo,
	enum CBLAS_TRANSPOSE transA, enum CBLAS_DIAG Diag, const matrix *A,
	vector *x)
{
        hipblasOperation_t tA;
        hipblasDiagType_t di;
        hipblasFillMode_t ul;

        if (A->order == CblasColMajor) {
                tA = (transA == CblasTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
                ul = (uplo == CblasLower) ?
                     HIPBLAS_FILL_MODE_LOWER : HIPBLAS_FILL_MODE_UPPER;
        } else {
                tA = (transA == CblasTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
                ul = (uplo == CblasLower) ?
                     HIPBLAS_FILL_MODE_UPPER : HIPBLAS_FILL_MODE_LOWER;
        }

        di = Diag == CblasNonUnit ? HIPBLAS_DIAG_NON_UNIT : HIPBLAS_DIAG_UNIT;

        if (!linalg_handle)
                return;

        CUBLAS(trsv)(*(hipblasHandle_t *) linalg_handle, ul, tA, di,
                (int) A->size1, A->data, (int) A->ld, x->data, (int) x->stride);
        CUDA_CHECK_ERR;
}


void blas_sbmv(void * linalg_handle, enum CBLAS_ORDER order,
	enum CBLAS_UPLO uplo, const size_t num_superdiag, const ok_float alpha,
	const vector * vecA, const vector * x, const ok_float beta, vector * y)
{
        hipblasFillMode_t ul;
        if (order == CblasRowMajor)
                ul = (uplo == CblasLower) ?
                     HIPBLAS_FILL_MODE_LOWER : HIPBLAS_FILL_MODE_UPPER;
        else
                ul = (uplo == CblasLower) ?
                     HIPBLAS_FILL_MODE_LOWER : HIPBLAS_FILL_MODE_UPPER;

        CUBLAS(sbmv)(*(hipblasHandle_t *) linalg_handle, ul,
               (int) y->size, (int) num_superdiag, &alpha,
               vecA->data, (int) (num_superdiag + 1),
               x->data, (int) x->stride, &beta, y->data, (int) y->stride);
}

void blas_diagmv(void * linalg_handle, const ok_float alpha,
        const vector * vecA, const vector * x, const ok_float beta, vector * y)
{
        blas_sbmv(linalg_handle, CblasColMajor, CblasLower, 0, alpha, vecA, x,
                beta, y);
}

/* BLAS LEVEL 3 */
void blas_syrk(void * linalg_handle, enum CBLAS_UPLO uplo,
	enum CBLAS_TRANSPOSE transA, ok_float alpha, const matrix * A,
	ok_float beta, matrix * C)
{

        hipblasOperation_t tA;
        hipblasFillMode_t ul;
        const int k = (transA == CblasNoTrans) ?
                      (int) A->size2 : (int) A->size1;

        if (A->order == CblasColMajor) {
                tA = (transA == CblasTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
                ul = (uplo == CblasLower) ?
                     HIPBLAS_FILL_MODE_LOWER : HIPBLAS_FILL_MODE_UPPER;
        } else {
                tA = (transA == CblasTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
                ul = (uplo == CblasLower) ?
                     HIPBLAS_FILL_MODE_UPPER : HIPBLAS_FILL_MODE_LOWER;
        }


        if (!linalg_handle)
                return;

        if ( !__matrix_order_compat(A, C, "A", "C", "blas_syrk") )
                return;


        CUBLAS(syrk)(*(hipblasHandle_t *) linalg_handle, ul, tA, (int) C->size2,
                k, &alpha, A->data, (int) A->ld, &beta, C->data, (int) C->ld);

        CUDA_CHECK_ERR;
}

void blas_gemm(void * linalg_handle, enum CBLAS_TRANSPOSE transA,
        enum CBLAS_TRANSPOSE transB, ok_float alpha, const matrix * A,
        const matrix * B, ok_float beta, matrix * C)
{
        hipblasOperation_t tA, tB;
        int s1, s2;

        const int k = (transA == CblasNoTrans) ?
                      (int) A->size2 : (int) A->size1;

        s1 = (A->order == CblasRowMajor) ? (int) C->size2 : (int) C->size1;
        s2 = (A->order == CblasRowMajor) ? (int) C->size1 : (int) C->size2;
        if (A->order == CblasColMajor) {
                tA = transA == CblasTrans ? HIPBLAS_OP_T : HIPBLAS_OP_N;
                tB = transB == CblasTrans ? HIPBLAS_OP_T : HIPBLAS_OP_N;
        } else {
                tA = transB == CblasTrans ? HIPBLAS_OP_T : HIPBLAS_OP_N;
                tB = transA == CblasTrans ? HIPBLAS_OP_T : HIPBLAS_OP_N;
        }

        if (!linalg_handle)
                return;

        if (!__matrix_order_compat(A, B, "A", "B", "blas_gemm") ||
                !__matrix_order_compat(A, C, "A", "C", "blas_gemm"))
                return;

        CUBLAS(gemm)(*(hipblasHandle_t *) linalg_handle, tA, tB, s1, s2, k,
                &alpha, A->data, (int) A->ld, B->data, (int) B->ld, &beta,
                C->data, (int) C->ld);

        CUDA_CHECK_ERR;
}


void blas_trsm(void * linalg_handle, enum CBLAS_SIDE Side, enum CBLAS_UPLO uplo,
        enum CBLAS_TRANSPOSE transA, enum CBLAS_DIAG Diag, ok_float alpha,
        const matrix *A, matrix *B)
{
        printf("Method `blas_trsm()` not implemented for GPU\n");
}

/*
 * LINEAR ALGEBRA routines
 * =======================
 */

/* cholesky decomposition of a single block */
__global__ void __block_chol(ok_float * A, uint iter, uint ld,
	enum CBLAS_ORDER ord)
{
        uint col, row, mat_dim, global_col, global_row, i;
        const uint kTileLD = kTileSize + 1u;
        __shared__ ok_float L[kTileLD * kTileSize];
        ok_float a11;

        col = threadIdx.x;
        row = threadIdx.y;
        mat_dim = blockDim.x;

        global_col = iter * kTileSize + col;
        global_row = iter * kTileSize + row;

        ok_float& (* get)(ok_float * A, uint i, uint j, uint stride) =
                (ord == CblasRowMajor) ? __matrix_get_r : __matrix_get_c;

        get(L, row, col, kTileLD) = get(A, global_row, global_col, ld);
        __syncthreads();


        for (i = 0; i < mat_dim; ++i) {
                /* l11 = sqrt(a11) */
                a11 = sqrt(get(L, i, i, kTileLD));
	        __syncthreads();


	        /* l21 = a21 / l11 */
	        if (row >= i && col == 0)
	                get(L, row, i, kTileLD) /= a11;
                __syncthreads();


	        /* a22 -= l21 * l21' */
	        if (row >= col && col > i)
	                get(L, row, col, kTileLD) -=
	                     get(L, col, i, kTileLD) *
	                     get(L, row, i, kTileLD);
                __syncthreads();
        }

        if (row >= col)
                get(A, global_row, global_col, ld) =
                get(L, row, col, kTileLD);
}

__global__ void __block_trsv(ok_float * A, uint iter, uint n, uint ld,
	enum CBLAS_ORDER ord)
{
        uint tile_idx, row, global_row, global_col, i, j;
        const uint kTileLD = kTileSize + 1u;
        __shared__ ok_float L[kTileLD * kTileSize];
        __shared__ ok_float A12[kTileLD * kTileSize];

        tile_idx = blockIdx.x;
        row = threadIdx.x;
        global_col = iter * kTileSize;
        global_row = iter * kTileSize + row;

        ok_float& (* get)(ok_float * A, uint i, uint j, uint stride) =
                (ord == CblasRowMajor) ? __matrix_get_r : __matrix_get_c;




        /* Load A -> L columnwise. */
        for (i = 0; i < kTileSize; ++i)
                get(L, row, i, kTileLD) = get(A, global_row, global_col + i,
                	ld);
        __syncthreads();

        global_row = row + (iter + tile_idx + 1u) * kTileSize;


        if (global_row < n)
	        for (i = 0; i < kTileSize; ++i)
	                get(A12, row, i, kTileLD) =
	        		get(A, global_row, global_col + i, ld);
        __syncthreads();

        if (global_row < n)
	        for (i = 0; i < kTileSize; ++i) {
	                for (j = 0; j < i; ++j)
	                        get(A12, row, i, kTileLD) -=
	                		get(A12, row, j, kTileLD) *
	                                get(L, i, j, kTileLD);
		        get(A12, row, i, kTileLD) /= get(L, i, i, kTileLD);
	        }
        __syncthreads();

        if (global_row < n)
	        for (uint i = 0; i < kTileSize; ++i)
	                get(A, global_row, global_col + i, ld) =
	        		get(A12, row, i, kTileLD);
        __syncthreads();
}

/*
 * Block Cholesky.
 *   l11 l11^T = a11
 *   l21 = a21 l11^(-T)
 *   a22 = a22 - l21 l21^T
 *
 * Stores result in Lower triangular part.
 */
void linalg_cholesky_decomp(void * linalg_handle, matrix * A)
{
        hipblasStatus_t err;
        hipStream_t stm;
        uint num_tiles, grid_dim, i;

        err = hipblasGetStream(*(hipblasHandle_t *) linalg_handle, &stm);
        num_tiles = (A->size1 + kTileSize - 1u) / kTileSize;

        for (i = 0; i < num_tiles; ++i) {
                if (err != HIPBLAS_STATUS_SUCCESS)
                        break;

                /* L11 = chol(A11) */
                uint block_dim_1d = kTileSize < A->size1 - i * kTileSize ? \
                                    kTileSize : A->size1 - i * kTileSize;
	        dim3 block_dim(block_dim_1d, block_dim_1d);

                __block_chol<<<1, block_dim, 0, stm>>>(A->data, i,
                	(uint) A->ld, A->order);
                CUDA_CHECK_ERR;

                if (i == num_tiles - 1u)
                        break;

                /* L21 = A21 * L21^-T */
                grid_dim = num_tiles - i - 1u;
                matrix L21 = matrix_submatrix_gen(A, (i + 1) * kTileSize,
                	i * kTileSize, A->size1 - (i + 1) * kTileSize,
                	kTileSize);

                __block_trsv<<<grid_dim, kTileSize, 0, stm>>>(A->data, i,
                        (uint) A->size1, (uint) A->ld, A->order);
                CUDA_CHECK_ERR;

                /* A22 -= L21 * L21^T */
                matrix A22 = matrix_submatrix_gen(A, (i + 1) * kTileSize,
                	(i + 1) * kTileSize, A->size1 - (i + 1) * kTileSize,
                        A->size1 - (i + 1) * kTileSize);

                blas_syrk(linalg_handle, CblasLower, CblasNoTrans, -kOne, &L21,
                        kOne, &A22);
        }
}


/* Cholesky solve */
void linalg_cholesky_svx(void * linalg_handle, const matrix * L, vector * x)
{
        blas_trsv(linalg_handle, CblasLower, CblasNoTrans, CblasNonUnit, L, x);
        blas_trsv(linalg_handle, CblasLower, CblasTrans, CblasNonUnit, L, x);
}

/* device reset */
ok_status ok_device_reset()
{
        hipDeviceReset();
        CUDA_CHECK_ERR;
        return OPTKIT_SUCCESS;
}


#ifdef __cplusplus
}
#endif