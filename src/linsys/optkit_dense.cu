#include "hip/hip_runtime.h"
#include "optkit_defs_gpu.h"
#include "optkit_dense.h"


/* row major data retrieval */
__device__ inline ok_float& __matrix_get_r(ok_float * A, uint i, uint j,
	uint stride)
{
	return A[i * stride + j];
}

/* column major data retrieval */
__device__ inline ok_float& __matrix_get_c(ok_float * A, uint i, uint j,
	uint stride)
{
	return A[i + j * stride];
}

#ifdef __cplusplus
extern "C" {
#endif

void denselib_version(int * maj, int * min, int * change, int * status)
{
	* maj = OPTKIT_VERSION_MAJOR;
	* min = OPTKIT_VERSION_MINOR;
	* change = OPTKIT_VERSION_CHANGE;
	* status = (int) OPTKIT_VERSION_STATUS;
}

/* cholesky decomposition of a single block */
static __global__ void __block_chol(ok_float * A, uint iter, uint ld,
	const enum CBLAS_ORDER ord)
{
	uint col, row, mat_dim, global_col, global_row, i;
	const uint kTileLD = kTileSize + 1u;
	__shared__ ok_float L[kTileLD * kTileSize];
	ok_float a11;

	ok_float& (* get)(ok_float * A, uint i, uint j, uint stride) =
		(ord == CblasRowMajor) ? __matrix_get_r : __matrix_get_c;

	col = threadIdx.x;
	row = threadIdx.y;
	mat_dim = blockDim.x;

	global_col = iter * kTileSize + col;
	global_row = iter * kTileSize + row;

	get(L, row, col, kTileLD) = get(A, global_row, global_col, ld);
	__syncthreads();

	for (i = 0; i < mat_dim; ++i) {
		/* l11 = sqrt(a11) */
		a11 = sqrt(get(L, i, i, kTileLD));
		__syncthreads();


		/* l21 = a21 / l11 */
		if (row >= i && col == 0)
			get(L, row, i, kTileLD) /= a11;
		__syncthreads();

		/* a22 -= l21 * l21' */
		if (row >= col && col > i)
			get(L, row, col, kTileLD) -=
			     get(L, col, i, kTileLD) *
			     get(L, row, i, kTileLD);
		__syncthreads();
	}

	if (row >= col)
		get(A, global_row, global_col, ld) = get(L, row, col, kTileLD);
}

static __global__ void __block_trsv(ok_float * A, uint iter, uint n, uint ld,
	const enum CBLAS_ORDER ord)
{
	uint tile_idx, row, global_row, global_col, i, j;
	const uint kTileLD = kTileSize + 1u;
	__shared__ ok_float L[kTileLD * kTileSize];
	__shared__ ok_float A12[kTileLD * kTileSize];

	ok_float& (* get)(ok_float * A, uint i, uint j, uint stride) =
		(ord == CblasRowMajor) ? __matrix_get_r : __matrix_get_c;

	tile_idx = blockIdx.x;
	row = threadIdx.x;
	global_col = iter * kTileSize;
	global_row = iter * kTileSize + row;

	/* Load A -> L columnwise. */
	for (i = 0; i < kTileSize; ++i)
		get(L, row, i, kTileLD) = get(A, global_row, global_col + i,
			ld);
	__syncthreads();

	global_row = row + (iter + tile_idx + 1u) * kTileSize;

	if (global_row < n)
		for (i = 0; i < kTileSize; ++i)
			get(A12, row, i, kTileLD) =
				get(A, global_row, global_col + i, ld);
	__syncthreads();

	if (global_row < n)
		for (i = 0; i < kTileSize; ++i) {
			for (j = 0; j < i; ++j)
				get(A12, row, i, kTileLD) -=
					get(A12, row, j, kTileLD) *
					get(L, i, j, kTileLD);
			get(A12, row, i, kTileLD) /= get(L, i, i, kTileLD);
		}
	__syncthreads();

	if (global_row < n)
		for (uint i = 0; i < kTileSize; ++i)
			get(A, global_row, global_col + i, ld) =
				get(A12, row, i, kTileLD);
	__syncthreads();
}

/*
 * Block Cholesky.
 *   l11 l11^T = a11
 *   l21 = a21 l11^(-T)
 *   a22 = a22 - l21 l21^T
 *
 * Stores result in Lower triangular part.
 */
ok_status linalg_cholesky_decomp(void * linalg_handle, matrix * A)
{
	hipblasStatus_t err;
	hipStream_t stm;
	uint num_tiles, grid_dim, i;
	matrix L21, A22;

	if (!linalg_handle)
		return OK_SCAN_ERR( OPTKIT_ERROR_UNALLOCATED );
	OK_CHECK_MATRIX(A);
	if (A->size1 != A->size2)
		return OK_SCAN_ERR( OPTKIT_ERROR_DIMENSION_MISMATCH );

	err = OK_SCAN_CUBLAS( hipblasGetStream(*(hipblasHandle_t *) linalg_handle,
		&stm) );
	num_tiles = (A->size1 + kTileSize - 1u) / kTileSize;

	for (i = 0; i < num_tiles && !err; ++i) {
		/* L11 = chol(A11) */
		uint block_dim_1d = kTileSize < A->size1 - i * kTileSize ? \
				    kTileSize : A->size1 - i * kTileSize;
		dim3 block_dim(block_dim_1d, block_dim_1d);

		if (!err)
			__block_chol<<<1, block_dim, 0, stm>>>(A->data, i,
				(uint) A->ld, A->order);
		hipDeviceSynchronize();
		OK_RETURNIF_ERR( OK_STATUS_CUDA )

		if (i == num_tiles - 1u)
			break;

		/* L21 = A21 * L21^-T */
		grid_dim = num_tiles - i - 1u;
		OK_RETURNIF_ERR( matrix_submatrix(&L21, A, (i + 1) * kTileSize,
			i * kTileSize, A->size1 - (i + 1) * kTileSize,
			kTileSize) );

		if (!err)
			__block_trsv<<<grid_dim, kTileSize, 0, stm>>>(A->data,
				i, (uint) A->size1, (uint) A->ld, A->order);
		hipDeviceSynchronize();
		OK_RETURNIF_ERR( OK_STATUS_CUDA )

		/* A22 -= L21 * L21^T */
		OK_RETURNIF_ERR( matrix_submatrix(&A22, A, (i + 1) * kTileSize,
			(i + 1) * kTileSize, A->size1 - (i + 1) * kTileSize,
			A->size1 - (i + 1) * kTileSize) );

		OK_RETURNIF_ERR( blas_syrk(linalg_handle, CblasLower,
			CblasNoTrans, -kOne, &L21, kOne, &A22) );
	}
	return err;
}

/* Cholesky solve */
ok_status linalg_cholesky_svx(void * linalg_handle, const matrix * L,
	vector * x)
{
	OK_CHECK_MATRIX(L);
	OK_CHECK_VECTOR(x);

	if (!linalg_handle)
		return OK_SCAN_ERR( OPTKIT_ERROR_UNALLOCATED );
	if (L->size1 != L->size2 || L->size1 != x->size)
		return OK_SCAN_ERR( OPTKIT_ERROR_DIMENSION_MISMATCH );

	OK_RETURNIF_ERR( err, blas_trsv(linalg_handle, CblasLower, CblasNoTrans,
		CblasNonUnit, L, x) );
	return blas_trsv(linalg_handle, CblasLower, CblasTrans,
		CblasNonUnit, L, x) );
}

static __global__ void __block_col_squares(const ok_float * A,
	const size_t size1, const size_t size2, const size_t row_stride,
	const size_t col_stride, ok_float * v, const size_t stride_v,
	const size_t i, const size_t j)
{
	uint col, row, global_col, global_row;
	const uint kTileLD = kTileSize + 1u;
	__shared__ ok_float Asub[kTileLD * kTileSize];
	__shared__ ok_float vsub[kTileSize];

	row = threadIdx.x;
	col = threadIdx.y;
	global_row = i * kTileSize + row;
	global_col = j * kTileSize + col;

	if (global_row >= size1 || global_col >= size2)
		return;

	Asub[row * kTileLD + col] = A[global_row * row_stride +
		global_col * col_stride];
	if (row == 0)
		vsub[row] = v[global_row * stride_v];
	__syncthreads();

	vsub[row] += Asub[row * kTileLD + col] * Asub[row * kTileLD + col];
	__syncthreads();

	v[global_row * stride_v] = vsub[row];
}

ok_status linalg_matrix_row_squares(const enum CBLAS_TRANSPOSE t,
	const matrix * A, vector * v)
{
	OK_CHECK_MATRIX(A);
	OK_CHECK_VECTOR(v);

	uint i, j;
	uint block_size = kTiles2D * kTileSize;

	dim3 grid_dim(kTileSize, kTileSize);
	dim3 blk_dim(kTiles2D, kTiles2D);

	int transpose = t == CblasTrans;
	int rowmajor = A->order == CblasRowMajor;

	/*
	 *	transpose: multiply A^T * A: work with columns of A
	 *	non-transpose: multiply A * A^T: work with rows of A
	 *		(columns of A^T)
	 */
	size_t size1 = (transpose) ? A->size1 : A->size2;
	size_t size2 = (transpose) ? A->size2 : A->size1;
	size_t row_stride = (transpose == rowmajor) ? A->ld : 1;
	size_t col_stride = (transpose == rowmajor) ? 1 : A->ld;

	if (v->size != size1)
		return OK_SCAN_ERR( OPTKIT_ERROR_DIMENSION_MISMATCH );

	/* transform one bundle of [block_size] columns x M rows per stream */
	for (j = 0; j < size2; j += block_size) {
		hipStream_t s;
		hipStreamCreate(&s);
		for (i = 0; i < size1; i += block_size)
			__block_col_squares<<<grid_dim, blk_dim, 0, s>>>(
				A->data, size1, size2, row_stride, col_stride,
				v->data, v->stride, i, j);
		hipStreamDestroy(s);
	}
	hipDeviceSynchronize();
	return OK_STATUS_CUDA;
}

static __device__ void __entry_add(ok_float * data, const size_t row,
	const size_t col, const size_t stride_r, const size_t stride_c,
	const ok_float value)
{
	data[row * stride_r + col * stride_c] += value;
}

static __device__ void __entry_mul(ok_float * data, const size_t row,
	const size_t col, const size_t stride_r, const size_t stride_c,
	const ok_float value)
{
	data[row * stride_r + col * stride_c] *= value;
}

static __global__ void __matrix_broadcast_vector(ok_float * A,
	const size_t size1, const size_t size2, const size_t row_stride,
	const size_t col_stride, const ok_float * v, const size_t stride_v,
	const size_t i, const size_t j,
	void (* inplace_op_)(ok_float * data, const size_t row,
	const size_t col, const size_t stride_r, const size_t stride_c,
	const ok_float value))
{
	uint col, row, global_col, global_row;
	const uint kTileLD = kTileSize + 1u;
	__shared__ ok_float Asub[kTileLD * kTileSize];
	__shared__ ok_float vsub[kTileSize];

	row = threadIdx.x;
	col = threadIdx.y;
	global_row = i * kTileSize + row;
	global_col = j * kTileSize + col;

	if (global_row >= size1 || global_col >= size2)
		return;

	Asub[row * kTileLD + col] = A[global_row * row_stride +
		global_col * col_stride];
	if (col == 0)
		vsub[row] = v[global_row * stride_v];
	__syncthreads();

	inplace_op_(Asub, row, col, kTileLD, 1, vsub[row]);
	__syncthreads();

	A[global_row * row_stride + global_col * col_stride] =
		Asub[row * kTileLD + col];
}

ok_status linalg_matrix_broadcast_vector(matrix * A, const vector * v,
	const enum OPTKIT_TRANSFORM operation, const enum CBLAS_SIDE side)
{
	OK_CHECK_MATRIX(A);
	OK_CHECK_VECTOR(v);

	uint i, j;
	uint block_size = kTiles2D * kTileSize;

	dim3 grid_dim(kTileSize, kTileSize);
	dim3 blk_dim(kTiles2D, kTiles2D);
	int left = side == CblasLeft;
	int rowmajor = A->order == CblasRowMajor;

	/*
	 * logic for row/col broadcast:
	 *	side = left: broadcast to each row of A^T
	 *	side = right: broadcast to each row of A
	 */
	size_t size1 = (left) ? A->size1 : A->size2;
	size_t size2 = (left) ? A->size2 : A->size1;
	size_t row_stride = (left == rowmajor) ? A->ld : 1;
	size_t col_stride = (left == rowmajor) ? 1 : A->ld;
	void (*transform)(ok_float * data, const size_t row, const size_t col,
		const size_t stride_r, const size_t stride_c,
		const ok_float value) = (operation == OkTransformScale) ?
		__entry_mul : __entry_add;

	if (size1 != v->size)
		return OK_SCAN_ERR( OPTKIT_ERROR_DIMENSION_MISMATCH );

	/* transform one bundle of [block_size] rows x N cols per stream */
	for (i = 0; i < size1; i += block_size) {
		hipStream_t s;
		hipStreamCreate(&s);
		for (j = 0; j < size2; j += block_size)
			__matrix_broadcast_vector<<<grid_dim, blk_dim, 0, s>>>(
				A->data, size1, size2, row_stride, col_stride,
				v->data, v->stride, i, j, transform);
		hipStreamDestroy(s);
	}
	hipDeviceSynchronize();
	return OK_STATUS_CUDA;
}

static __global__ void __matrix_row_indmin(size_t * indmin,
	const size_t stride_indmin, ok_float * minima, const size_t stride_min,
	const ok_float * A, const size_t size1, const size_t size2,
	const size_t row_stride, const size_t col_stride, const size_t i,
	const size_t j)
{
	uint row = threadIdx.x;
	uint col = threadIdx.y;
	uint global_row = i + row;
	uint global_col = j + col;
	const uint kTileLD = kTileSize + 1u;
	__shared__ ok_float Asub[kTileLD * kTileSize];
	__shared__ ok_float minsub[kTileSize];
	__shared__ size_t indminsub[kTileSize];
	ok_float previous;

	if (global_row >= size1 || global_col >= size2)
		return;

	Asub[row * kTileLD + col] = A[global_row * row_stride +
		global_col * col_stride];
	__syncthreads();

	if (col == 0)
		minsub[row] = MATH(fmin)(minima[global_row * stride_min],
			OK_FLOAT_MAX);
	__syncthreads();

	previous = minsub[row];
	minsub[row] = MATH(fmin)(minsub[row], Asub[row * kTileLD + col]);
	if (minsub[row] != previous)
		indminsub[row] = global_col;
	__syncthreads();

	if (col == 0)
		minima[global_row * stride_min] = minsub[row];
	__syncthreads();

	if (col == 0)
		indmin[global_row * stride_indmin] = indminsub[row];
}

static __global__ void __matrix_row_reduce(ok_float * reduced,
	const size_t stride, const ok_float * A, const size_t size1,
	const size_t size2, const size_t row_stride, const size_t col_stride,
	const size_t i, const size_t j, const ok_float default_value,
	ok_float (* binary_op_)(const ok_float first, const ok_float second))
{
	uint row = threadIdx.x;
	uint col = threadIdx.y;
	uint global_row = i + row;
	uint global_col = j + col;
	const uint kTileLD = kTileSize + 1u;
	__shared__ ok_float Asub[kTileLD * kTileSize];
	__shared__ ok_float reduced_sub[kTileSize];

	if (global_row >= size1 || global_col >= size2)
		return;

	Asub[row * kTileLD + col] = A[global_row * row_stride +
		global_col * col_stride];
	__syncthreads();

	if (col == 0)
		reduced_sub[row] = binary_op_(reduced[global_row * stride],
			default_value);
	__syncthreads();

	reduced_sub[row] = binary_op_(reduced_sub[row],
		Asub[row * kTileLD + col]);
	__syncthreads();

	if (col == 0)
		reduced[global_row * stride] = reduced_sub[row];
}

ok_status linalg_matrix_reduce_indmin(indvector * indices, vector * minima,
	matrix * A, const enum CBLAS_SIDE side)
{
	OK_CHECK_VECTOR(indices);
	OK_CHECK_VECTOR(minima);
	OK_CHECK_MATRIX(A);

	uint i, j;
	uint block_size = kTiles2D * kTileSize;

	dim3 grid_dim(kTileSize, kTileSize);
	dim3 blk_dim(kTiles2D, kTiles2D);
	int left = side == CblasLeft;
	int rowmajor = A->order == CblasRowMajor;

	/*
	 * logic for row/col reduction:
	 *	side = left: reduce each row of A^T 	(analoguous to A^T * 1)
	 *	side = right: reduce each row of A 	(analogous to A * 1)
	 */
	size_t size1 = (left) ? A->size1 : A->size2;
	size_t size2 = (left) ? A->size2 : A->size1;
	size_t row_stride = (left == rowmajor) ? A->ld : 1;
	size_t col_stride = (left == rowmajor) ? 1 : A->ld;

	if (size2 != minima->size || indices->size != minima->size)
		return OK_SCAN_ERR( OPTKIT_ERROR_DIMENSION_MISMATCH );

	OK_RETURNIF_ERR( vector_set_all(minima, OK_FLOAT_MAX) );

	/* reduce one bundle of [block_size] rows x N cols per stream */
	for (i = 0; i < size1; i += block_size) {
		hipStream_t s;
		hipStreamCreate(&s);
		for (j = 0; j < size2; j += block_size)
			__matrix_row_indmin<<<grid_dim, blk_dim, 0, s>>>(
				indices->data, indices->stride, minima->data,
				minima->stride, A->data, size1, size2,
				row_stride, col_stride, i, j);
		hipStreamDestroy(s);
	}
	hipDeviceSynchronize();
	return OK_STATUS_CUDA;
}

static ok_status __matrix_reduce_binary(vector * reduced, matrix * A,
	const enum CBLAS_SIDE side, const ok_float default_value,
	ok_float (* binary_op_)(const ok_float first, const ok_float second))
{
	OK_CHECK_VECTOR(reduced);
	OK_CHECK_MATRIX(A);

	uint i, j;
	uint block_size = kTiles2D * kTileSize;

	dim3 grid_dim(kTileSize, kTileSize);
	dim3 blk_dim(kTiles2D, kTiles2D);
	int left = side == CblasLeft;
	int rowmajor = A->order == CblasRowMajor;

	/*
	 * logic for row/col reduction:
	 *	side = left: reduce each row of A^T 	(analoguous to A^T * 1)
	 *	side = right: reduce each row of A 	(analogous to A * 1)
	 */
	size_t size1 = (left) ? A->size1 : A->size2;
	size_t size2 = (left) ? A->size2 : A->size1;
	size_t row_stride = (left == rowmajor) ? A->ld : 1;
	size_t col_stride = (left == rowmajor) ? 1 : A->ld;

	if (size2 != reduced->size)
		return OK_SCAN_ERR( OPTKIT_ERROR_DIMENSION_MISMATCH );

	OK_RETURNIF_ERR( vector_set_all(reduced, default_value) );

	/* reduce one bundle of [block_size] rows x N cols per stream */
	for (i = 0; i < size1; i += block_size) {
		hipStream_t s;
		hipStreamCreate(&s);
		for (j = 0; j < size2; j += block_size)
			__matrix_row_reduce<<<grid_dim, blk_dim, 0, s>>>(
				reduced->data, reduced->stride, A->data, size1,
				size2, row_stride, col_stride, i, j,
				default_value, binary_op_);
		hipStreamDestroy(s);
	}
	hipDeviceSynchronize();
	return OK_STATUS_CUDA;
}

ok_status linalg_matrix_reduce_min(vector * minima, matrix * A,
	const enum CBLAS_SIDE side)
{
	return __matrix_reduce_binary(minima, A, side, OK_FLOAT_MAX,
		MATH(fmin));
}

ok_status linalg_matrix_reduce_max(vector * maxima, matrix * A,
	const enum CBLAS_SIDE side)
{
	return __matrix_reduce_binary(maxima, A, side, -OK_FLOAT_MAX,
		MATH(fmax));
}

#ifdef __cplusplus
}
#endif
