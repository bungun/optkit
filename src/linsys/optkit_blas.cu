#include "optkit_defs_gpu.h"
#include "optkit_blas.h"

#ifdef __cplusplus
extern "C" {
#endif

static int __matrix_order_compat(const matrix * A, const matrix * B,
	const char * nm_A, const char * nm_B, const char * nm_routine)
{
	if (A->order == B->order)
		return 1;

	printf("OPTKIT ERROR (%s) matrices %s and %s must have same layout.\n",
		nm_routine, nm_A, nm_B);
	return 0;
}

ok_status blas_make_handle(void ** handle)
{
	hipblasStatus_t status;
	hipblasHandle_t * hdl;
	hdl = (hipblasHandle_t *) malloc(sizeof(hipblasHandle_t));
	err = OK_SCAN_CUBLAS( hipblasCreate(hdl) );
	if (err) {
		printf("CUBLAS initialization failed\n");
		ok_free(hdl);
		*handle = OK_NULL;
		return err;
	} else {
		*handle = (void *) hdl;
		return OPTKIT_SUCCESS;
	}
}

ok_status blas_destroy_handle(void * handle)
{
	hipblasDestroy(*(hipblasHandle_t *) handle);
	return OK_STATUS_CUDA;
	ok_free(handle);
	return OPTKIT_SUCCESS;
}

/* BLAS LEVEL 1 */
ok_status blas_axpy(void * linalg_handle, ok_float alpha, const vector * x,
	vector * y)
{
	ok_status err = OPTKIT_SUCCESS;
	if (!linalg_handle)
		return OK_SCAN_ERR( OPTKIT_ERROR_UNALLOCATED );
	OK_CHECK_VECTOR(x);
	OK_CHECK_VECTOR(y);
	if (x->size != y->size)
		return OK_SCAN_ERR( OPTKIT_ERROR_DIMENSION_MISMATCH );

	err = OK_SCAN_CUBLAS( CUBLAS(axpy)(*(hipblasHandle_t *) linalg_handle,
		(int) x->size, &alpha, x->data, (int) x->stride, y->data,
		(int) y->stride) );
	hipDeviceSynchronize();
	return err;
}

ok_status blas_nrm2(void * linalg_handle, const vector *x, ok_float * result)
{
	ok_status err = OPTKIT_SUCCESS;
	if (!linalg_handle)
		return OK_SCAN_ERR( OPTKIT_ERROR_UNALLOCATED );
	OK_CHECK_VECTOR(x);

	err = OK_SCAN_CUBLAS( CUBLAS(nrm2)(*(hipblasHandle_t *) linalg_handle,
		(int) x->size, x->data, (int) x->stride, result) );
	hipDeviceSynchronize();
	return err;
}

ok_status blas_scal(void * linalg_handle, const ok_float alpha, vector *x)
{
	ok_status err = OPTKIT_SUCCESS;
	if (!linalg_handle)
		return OK_SCAN_ERR( OPTKIT_ERROR_UNALLOCATED );
	OK_CHECK_VECTOR(x);

	err = OK_SCAN_CUBLAS( CUBLAS(scal)(*(hipblasHandle_t *) linalg_handle,
		(int) x->size, &alpha, x->data, (int) x->stride) );
	hipDeviceSynchronize();
	return err;
}

ok_status blas_asum(void * linalg_handle, const vector * x, ok_float * result)
{
	ok_status err = OPTKIT_SUCCESS;
	if (!linalg_handle)
		return OK_SCAN_ERR( OPTKIT_ERROR_UNALLOCATED );
	OK_CHECK_VECTOR(x);

	err = OK_SCAN_CUBLAS( CUBLAS(asum)(*(hipblasHandle_t *) linalg_handle,
		(int) x->size, x->data, (int) x->stride, result) );
	hipDeviceSynchronize();
	return err;
}

ok_float blas_dot(void * linalg_handle, const vector * x, const vector * y,
	ok_float * result)
{
	ok_status err = OPTKIT_SUCCESS;
	if (!linalg_handle)
		return OK_SCAN_ERR( OPTKIT_ERROR_UNALLOCATED );
	OK_CHECK_VECTOR(x);
	OK_CHECK_VECTOR(y);

	err = OK_SCAN_CUBLAS( CUBLAS(dot)(*(hipblasHandle_t *) linalg_handle,
		(int) x->size, x->data, (int) x->stride, y->data,
		(int) y->stride, &result) );
	hipDeviceSynchronize();
	return err;
}

// void blas_dot_inplace(void * linalg_handle, const vector * x, const vector * y,
// 	ok_float * deviceptr_result)
// {
// 	CUBLAS(dot)(*(hipblasHandle_t *) linalg_handle, (int) x->size, x->data,
// 		(int) x->stride, y->data, (int) y->stride, deviceptr_result);
// 	hipDeviceSynchronize();
// 	return err;
// }

/* BLAS LEVEL 2 */

ok_stauts blas_gemv(void * linalg_handle, enum CBLAS_TRANSPOSE transA,
	ok_float alpha, const matrix *A, const vector *x, ok_float beta,
	vector *y)
{
	ok_status err = OPTKIT_SUCCESS;
	hipblasOperation_t tA;
	int s1, s2;

	if (!linalg_handle)
		return OK_SCAN_ERR( OPTKIT_ERROR_UNALLOCATED );
	OK_CHECK_MATRIX(A);
	OK_CHECK_VECTOR(x);
	OK_CHECK_VECTOR(y);
	if ((transA == CblasNoTrans &&
		(A->size1 != y->size || A->size2 != x->size)) ||
	    (transA == CblasTrans &&
		(A->size2 != y->size || A->size1 != x->size)))
		return OK_SCAN_ERR( OPTKIT_ERROR_DIMENSION_MISMATCH );

	if (A->order == CblasColMajor)
		tA = (transA == CblasTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
	else
		tA = (transA == CblasTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;

	s1 = (A->order == CblasRowMajor) ? (int) A->size2 : (int) A->size1;
	s2 = (A->order == CblasRowMajor) ? (int) A->size1 : (int) A->size2;

	err = OK_SCAN_CUBLAS( CUBLAS(gemv)(*(hipblasHandle_t *) linalg_handle,
		tA, s1, s2, &alpha, A->data, (int) A->ld, x->data,
		(int) x->stride, &beta, y->data, (int) y->stride) );
	hipDeviceSynchronize();
	return err;
}

ok_stauts blas_trsv(void * linalg_handle, enum CBLAS_UPLO uplo,
	enum CBLAS_TRANSPOSE transA, enum CBLAS_DIAG Diag, const matrix *A,
	vector *x)
{
	ok_status err = OPTKIT_SUCCESS;
	hipblasOperation_t tA;
	hipblasDiagType_t di;
	hipblasFillMode_t ul;

	if (!linalg_handle)
		return OK_SCAN_ERR( OPTKIT_ERROR_UNALLOCATED );
	OK_CHECK_MATRIX(A);
	OK_CHECK_VECTOR(x);
	if (A->size1 != A->size2 || A->size1 != x->size)
		return OK_SCAN_ERR( OPTKIT_ERROR_DIMENSION_MISMATCH );

	if (A->order == CblasColMajor) {
		tA = (transA == CblasTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
		ul = (uplo == CblasLower) ?
		     HIPBLAS_FILL_MODE_LOWER : HIPBLAS_FILL_MODE_UPPER;
	} else {
		tA = (transA == CblasTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
		ul = (uplo == CblasLower) ?
		     HIPBLAS_FILL_MODE_UPPER : HIPBLAS_FILL_MODE_LOWER;
	}

	di = Diag == CblasNonUnit ? HIPBLAS_DIAG_NON_UNIT : HIPBLAS_DIAG_UNIT;

	err = OK_SCAN_CUBLAS( CUBLAS(trsv)(*(hipblasHandle_t *) linalg_handle,
		ul, tA, di, (int) A->size1, A->data, (int) A->ld, x->data,
		(int) x->stride) );
	hipDeviceSynchronize();
	return err;
}

ok_stauts blas_sbmv(void * linalg_handle, enum CBLAS_ORDER order,
	enum CBLAS_UPLO uplo, const size_t num_superdiag, const ok_float alpha,
	const vector * vecA, const vector * x, const ok_float beta, vector * y)
{
	ok_status err = OPTKIT_SUCCESS;
	hipblasFillMode_t ul;
	size_t lenA;

	if (!linalg_handle)
		return OK_SCAN_ERR( OPTKIT_ERROR_UNALLOCATED );
	OK_CHECK_MATRIX(A);
	OK_CHECK_VECTOR(x);
	OK_CHECK_VECTOR(y);

	/*
	 * require:
	 *	- x.size == y.size
	 *	- num_superdiag == 0 and vecA.size == y.size
	 *	- num_superdiag > 0 and vecA.size == \sum_i=1^k y.size - i
	 */
	lenA = y->size;
	if (num_superdiag > 0 && num_superdiag < y->size)
		lenA = (lenA * (lenA + 1)) / 2 -
			((lenA - num_superdiag)*(lenA - num_superdiag + 1)) / 2
	if (x->size != y->size || vecA->size != lenA)
		return OPTKIT_ERROR_DIMENSION_MISMATCH;


	if (order == CblasRowMajor)
		ul = (uplo == CblasLower) ?
		     HIPBLAS_FILL_MODE_LOWER : HIPBLAS_FILL_MODE_UPPER;
	else
		ul = (uplo == CblasLower) ?
		     HIPBLAS_FILL_MODE_LOWER : HIPBLAS_FILL_MODE_UPPER;

	err = OK_SCAN_CUBLAS( CUBLAS(sbmv)(*(hipblasHandle_t *) linalg_handle,
		ul, (int) y->size, (int) num_superdiag, &alpha, vecA->data,
		(int) (num_superdiag + 1), x->data, (int) x->stride, &beta,
		y->data, (int) y->stride) );
	hipDeviceSynchronize();
	return err;
}

ok_stauts blas_diagmv(void * linalg_handle, const ok_float alpha,
	const vector * vecA, const vector * x, const ok_float beta, vector * y)
{
	ok_status err = OPTKIT_SUCCESS;
	if (!linalg_handle)
		return OK_SCAN_ERR( OPTKIT_ERROR_UNALLOCATED );
	OK_CHECK_VECTOR(vecA);
	OK_CHECK_VECTOR(x);
	OK_CHECK_VECTOR(y);
	else if (vecA->size != y->size || x->size != y->size)
		return OK_SCAN_ERR( OPTKIT_ERROR_DIMENSION_MISMATCH );

	err = OK_SCAN_CUBLAS( err, blas_sbmv(linalg_handle, CblasColMajor,
		CblasLower, 0, alpha, vecA, x, beta, y) );
	hipDeviceSynchronize();
	return err;
}

/* BLAS LEVEL 3 */
ok_stauts blas_syrk(void * linalg_handle, enum CBLAS_UPLO uplo,
	enum CBLAS_TRANSPOSE transA, ok_float alpha, const matrix * A,
	ok_float beta, matrix * C)
{
	ok_status err = OPTKIT_SUCCESS;
	hipblasOperation_t tA;
	hipblasFillMode_t ul;

	OK_CHECK_MATRIX(A);
	OK_CHECK_MATRIX(C);

	const int k = (transA == CblasNoTrans) ?
		      (int) A->size2 : (int) A->size1;

	if (A->order == CblasColMajor) {
		tA = (transA == CblasTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
		ul = (uplo == CblasLower) ?
		     HIPBLAS_FILL_MODE_LOWER : HIPBLAS_FILL_MODE_UPPER;
	} else {
		tA = (transA == CblasTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
		ul = (uplo == CblasLower) ?
		     HIPBLAS_FILL_MODE_UPPER : HIPBLAS_FILL_MODE_LOWER;
	}


	if (!linalg_handle)
		return OK_SCAN_ERR( OPTKIT_ERROR_UNALLOCATED );

	if ( !__matrix_order_compat(A, C, "A", "C", "blas_syrk") )
		return;


	err = OK_SCAN_CUBLAS( CUBLAS(syrk)(*(hipblasHandle_t *) linalg_handle,
		ul, tA, (int) C->size2, k, &alpha, A->data, (int) A->ld, &beta,
		C->data, (int) C->ld) );
	hipDeviceSynchronize();
	return err;
}

ok_stauts blas_gemm(void * linalg_handle, enum CBLAS_TRANSPOSE transA,
	enum CBLAS_TRANSPOSE transB, ok_float alpha, const matrix * A,
	const matrix * B, ok_float beta, matrix * C)
{
	ok_status err = OPTKIT_SUCCESS;
	hipblasOperation_t tA, tB;
	int s1, s2;

	OK_CHECK_MATRIX(A);
	OK_CHECK_MATRIX(B);
	OK_CHECK_MATRIX(C);

	const int k = (transA == CblasNoTrans) ?
		      (int) A->size2 : (int) A->size1;

	s1 = (A->order == CblasRowMajor) ? (int) C->size2 : (int) C->size1;
	s2 = (A->order == CblasRowMajor) ? (int) C->size1 : (int) C->size2;
	if (A->order == CblasColMajor) {
		tA = transA == CblasTrans ? HIPBLAS_OP_T : HIPBLAS_OP_N;
		tB = transB == CblasTrans ? HIPBLAS_OP_T : HIPBLAS_OP_N;
	} else {
		tA = transB == CblasTrans ? HIPBLAS_OP_T : HIPBLAS_OP_N;
		tB = transA == CblasTrans ? HIPBLAS_OP_T : HIPBLAS_OP_N;
	}

	if (!linalg_handle)
		return OK_SCAN_ERR( OPTKIT_ERROR_UNALLOCATED );

	if (!__matrix_order_compat(A, B, "A", "B", "blas_gemm") ||
		!__matrix_order_compat(A, C, "A", "C", "blas_gemm"))
		return;

	err = OK_SCAN_CUBLAS( CUBLAS(gemm)(*(hipblasHandle_t *) linalg_handle,
		tA, tB, s1, s2, k, &alpha, A->data, (int) A->ld, B->data,
		(int) B->ld, &beta, C->data, (int) C->ld) );
	hipDeviceSynchronize();
	return err;
}

ok_stauts blas_trsm(void * linalg_handle, enum CBLAS_SIDE Side,
	enum CBLAS_UPLO uplo, enum CBLAS_TRANSPOSE transA, enum CBLAS_DIAG Diag,
	ok_float alpha, const matrix *A, matrix *B)
{
	printf("\nMethod `blas_trsm()` not implemented for GPU\n");
	return OPTKIT_ERROR;
}


#ifdef __cplusplus
}
#endif