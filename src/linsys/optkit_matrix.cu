#include "hip/hip_runtime.h"
#include "optkit_defs_gpu.h"
#include "optkit_matrix.h"

/*
 * MATRIX CUDA helper methods
 * ==========================
 */

template<typename T>
static __global__ void __strided_memcpy(T * x, size_t stride_x, const T * y,
	size_t stride_y, size_t size)
{
	uint i, tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (i = tid; i < size; i += gridDim.x * blockDim.x)
	x[i * stride_x] = y[i * stride_y];
}

/* row major setter */
template<typename T>
static __global__ void __matrix_set_r(T * data, T x, size_t stride, size_t size1,
	size_t size2)
{
	uint i, j;
	uint thread_id_row = blockIdx.x * blockDim.x + threadIdx.x;
	uint thread_id_col = blockIdx.y * blockDim.y + threadIdx.y;
	uint incr_x = gridDim.x * blockDim.x;
	uint incr_y = gridDim.y * blockDim.y;
	for (i = thread_id_row; i < size1; i += incr_x)
		for (j = thread_id_col; j < size2; j += incr_y)
			data[i * stride + j] = x;
}

/* column major setter */
template<typename T>
static __global__ void __matrix_set_c(T * data, T x, size_t stride,
	size_t size1, size_t size2)
{
	uint i, j;
	uint thread_id_row = blockIdx.x * blockDim.x + threadIdx.x;
	uint thread_id_col = blockIdx.y * blockDim.y + threadIdx.y;
	uint incr_x = gridDim.x * blockDim.x;
	uint incr_y = gridDim.y * blockDim.y;
	for (j = thread_id_col; j < size2; j += incr_y)
		for (i = thread_id_row; i < size1; i += incr_x)
			data[i + j * stride] = x;
}

template<typename T>
static void __matrix_set_all(matrix_<T> * A, T x)
{
	uint grid_dimx = calc_grid_dim(A->size1);
	uint grid_dimy = calc_grid_dim(A->size2);
	dim3 grid_dim(grid_dimx, grid_dimy, 1u);
	dim3 block_dim(kBlockSize2D, kBlockSize2D, 1u);

	if (A->order == CblasRowMajor)
		__matrix_set_r<T><<<grid_dim, block_dim>>>(A->data, x, A->ld,
			A->size1, A->size2);
	else
		__matrix_set_c<T><<<grid_dim, block_dim>>>(A->data, x, A->ld,
			A->size1, A->size2);
	hipDeviceSynchronize();
	CUDA_CHECK_ERR;
}

template<typename T>
static __global__ void __matrix_add_constant_diag(T * data, T x,
	size_t stride)
{
	uint i = blockIdx.x * blockDim.x + threadIdx.x;
	data[i * stride + i] += x;
}

template<typename T>
void matrix_alloc_(matrix_<T> * A, size_t m, size_t n, enum CBLAS_ORDER ord)
{
	A->size1 = m;
	A->size2 = n;
	ok_alloc_gpu(A->data, m * n * sizeof(ok_float));
	A->ld = (ord == CblasRowMajor) ? n : m;
	A->order = ord;
}

template<typename T>
void matrix_calloc_(matrix_<T> * A, size_t m, size_t n, enum CBLAS_ORDER ord)
{
	if (!A)
		return;
	matrix_alloc(A, m, n, ord);
	hipMemset(A->data, 0, m * n * sizeof(ok_float));
	CUDA_CHECK_ERR;
}

template<typename T>
void matrix_free_(matrix_<T> * A)
{
	if (!A || !A->data)
		return;
	ok_free_gpu(A->data);
	A->size1 = (size_t) 0;
	A->size2 = (size_t) 0;
	A->ld = (size_t) 0;

}

template<typename T>
void matrix_submatrix_(matrix_<T> * A_sub, matrix_<T> * A, size_t i, size_t j,
	size_t n1, size_t n2)
{
	if (!A_sub || !A)
		return;
	A_sub->size1 = n1;
	A_sub->size2 = n2;
	A_sub->ld = A->ld;
	A_sub->data = (A->order == CblasRowMajor) ?
		      A->data + (i * A->ld) + j : A->data + i + (j * A->ld);
	A_sub->order = A->order;
}

template<typename T>
void matrix_row_(vector_<T> * row, matrix_<T> * A, size_t i)
{
	if (!row || !A)
		return;
	row->size = A->size2;
	row->stride = (A->order == CblasRowMajor) ? 1 : A->ld;
	row->data = (A->order == CblasRowMajor) ?
		    A->data + (i * A->ld) : A->data + i;
}

template<typename T>
void matrix_column_(vector * col, matrix *A, size_t j)
{
	if (!col || !A)
		return;
	col->size = A->size1;
	col->stride = (A->order == CblasRowMajor) ? A->ld : 1;
	col->data = (A->order == CblasRowMajor) ?
		    A->data + j : A->data + (j * A->ld);
}

template<typename T>
void matrix_diagonal_(vector_<T> * diag, matrix_<T> *A)
{
	if (!diag || !A)
		return;
	diag->data = A->data;
	diag->stride = A->ld + 1;
	diag->size = (size_t) (A->size1 <= A->size2) ? A->size1 : A->size2;
}

template<typename T>
void matrix_cast_vector_(vector_<T> * v, matrix_<T> * A)
{
	if (!v || !A)
		return;
	v->size = A->size1 * A->size2;
	v->stride = 1;
	v->data = A->data;
}

template<typename T>
void matrix_view_array_(matrix_<T> * A, const T * base, size_t n1, size_t n2,
	enum CBLAS_ORDER ord)
{
	if (!A || !base)
		return;
	A->size1 = n1;
	A->size2 = n2;
	A->data = (ok_float *) base;
	A->ld = (ord == CblasRowMajor) ? n2 : n1;
	A->order = ord;
}

template<typename T>
void matrix_set_all_(matrix_<T> * A, T x)
{
	if (!A)
		return;
	__matrix_set_all<T>(A, x);
}

template<typename T>
void matrix_memcpy_mm_(matrix_<T> * A, const matrix_<T> * B)
{
	uint i, j, grid_dim;
	if (A->size1 != B->size1) {
		printf("error: m-dimensions must match for matrix memcpy\n");
		return;
	} else if (A->size2 != B->size2) {
		printf("error: n-dimensions must match for matrix memcpy\n");
		return;
	}

	if (A->order == B->order) {
		ok_memcpy_gpu(A->data, B->data,
			      A->size1 * A->size2 * sizeof(T));
	} else if (A->order == CblasRowMajor) {
		/* A row major, B column major */
		grid_dim = calc_grid_dim(A->size1);
		for (i = 0; i < A->size1; ++i) {
			__strided_memcpy<T><<<grid_dim, kBlockSize>>>(
				A->data + i * A->size2, 1,
				B->data + i, A->ld, A->size2);
			hipDeviceSynchronize();
		}
	} else {
		/* A column major, B row major */
		grid_dim = calc_grid_dim(A->size2);
		for (j= 0; j < A->size2; ++j) {
			__strided_memcpy<T><<<grid_dim, kBlockSize>>>(
				A->data + j * A->size1, 1,
				B->data + j, A->ld, A->size1);
			hipDeviceSynchronize();
		}
	}
	CUDA_CHECK_ERR;
}

/*
 * Handle the following cases:
 *      A->order = ord
 *      A->order != ord, ord == CblasColMajor (A row major, B col major)
 *      A->order != ord, ord == CblasRowMajor (A col major, B row major)
 */
template<typename T>
void matrix_memcpy_ma_(matrix_<T> * A, const T * B, const enum CBLAS_ORDER ord)
{
	uint i, j, grid_dim;
	T * row, * col;

	if (A->order == ord) {
		if (ord == CblasRowMajor)
			for (i = 0; i < A->size1; ++i)
				ok_memcpy_gpu(A->data + i * A->ld,
					B + i * A->size2, A->size2 * sizeof(T));
		else
			for (j = 0; j < A->size2; ++j)
				ok_memcpy_gpu(A->data + j * A->ld,
					B + j * A->size1, A->size1 * sizeof(T));
	} else if (ord == CblasColMajor) {
		ok_alloc_gpu(col, A->size1 * sizeof(ok_float));
		grid_dim = calc_grid_dim(A->size1);
		for (j = 0; j < A->size2; ++j) {
			ok_memcpy_gpu(col, B + j * A->size1,
				A->size1 * sizeof(T));
			__strided_memcpy<T><<<grid_dim, kBlockSize>>>(
				A->data + j, A->ld, col, 1, A->size1);
			hipDeviceSynchronize();
		}
		ok_free_gpu(col);
	} else {
		ok_alloc_gpu(row, A->size2 * sizeof(ok_float));
		grid_dim = calc_grid_dim(A->size2);
		for (i = 0; i < A->size1; ++i) {
			ok_memcpy_gpu(row, B + i * A->size2,
				      A->size2 * sizeof(T));
			__strided_memcpy<T><<<grid_dim, kBlockSize>>>(
				A->data + i, A->ld, row, 1, A->size2);
			hipDeviceSynchronize();
		}
		ok_free_gpu(row);
	}
	CUDA_CHECK_ERR;
}

/*
 * Handle the following cases:
 *      ord = B->order
 *      ord != B->order, ord == CblasRowMajor (A row major, B col major)
 *      ord != B->ord, order == CblasColMajor (A col major, B row major)
 */
template<typename T>
void matrix_memcpy_am_(T * A, const matrix_<T> * B, const enum CBLAS_ORDER ord)
{
	uint i, j, grid_dim;
	T * row, * col;
	if (ord == B->order) {
		if (ord == CblasRowMajor)
			for (i = 0; i < B->size1; ++i)
				ok_memcpy_gpu(A + i * B->size2,
					B->data + i * B->ld,
					B->size2 * sizeof(T));
		else
			for (j = 0; j < B->size2; ++j)
				ok_memcpy_gpu(A + j * B->size1,
					B->data + j * B->ld,
					B->size1 * sizeof(T));
	} else if (ord == CblasRowMajor) {
		ok_alloc_gpu(row, B->size2 * sizeof(ok_float));
		grid_dim = calc_grid_dim(B->size2);
		for (i = 0; i < B->size1; ++i) {
			__strided_memcpy<T><<<grid_dim, kBlockSize>>>(row, 1,
				B->data + i, B->ld, B->size2);
			hipDeviceSynchronize();
			ok_memcpy_gpu(A + i * B->size2, row,
				      B->size2 * sizeof(T));
		}
		ok_free_gpu(row);
	} else {
		ok_alloc_gpu(col, B->size1 * sizeof(ok_float));
		grid_dim = calc_grid_dim(B->size1);
		for (j = 0; j < B->size2; ++j) {
			__strided_memcpy<T><<<grid_dim, kBlockSize>>>(col, 1,
				B->data + j, B->ld, B->size1);
			hipDeviceSynchronize();
			ok_memcpy_gpu(A + j * B->size1, col,
				      B->size1 * sizeof(T));
		}
		ok_free_gpu(col);
	}
	CUDA_CHECK_ERR;
}

#ifdef __cplusplus
extern "C" {
#endif

void matrix_alloc(matrix * A, size_t m, size_t n, enum CBLAS_ORDER ord)
	{ matrix_alloc_<ok_float>(A, m, n, ord); }

void matrix_calloc(matrix * A, size_t m, size_t n, enum CBLAS_ORDER ord)
	{ matrix_calloc_<ok_float>(A, m, n, ord); }

void matrix_free(matrix * A)
	{ matrix_free_<ok_float>(A); }

void matrix_submatrix(matrix * A_sub, matrix * A, size_t i, size_t j, size_t n1,
	size_t n2)
	{ matrix_submatrix_<ok_float>(A_sub, A, i, j, n1, n2); }

void matrix_row(vector * row, matrix * A, size_t i)
	{ matrix_row_<ok_float>(row, A, i); }

void matrix_column(vector * col, matrix * A, size_t j)
	{ matrix_column_<ok_float>(col, A, j); }

void matrix_diagonal(vector * diag, matrix * A)
	{ matrix_diagonal_<ok_float>(diag, A); }

void matrix_cast_vector(vector * v, matrix * A)
	{ matrix_cast_vector_<ok_float>(v, A); }

void matrix_view_array(matrix * A, const ok_float * base, size_t n1, size_t n2,
        enum CBLAS_ORDER ord)
	{ matrix_view_array_<ok_float>(A, base, n1, n2, ord); }

void matrix_set_all(matrix * A, ok_float x)
	{ matrix_set_all_<ok_float>(A, x); }

void matrix_memcpy_mm(matrix * A, const matrix * B)
	{ matrix_memcpy_mm_<ok_float>(A, B); }

void matrix_memcpy_ma(matrix * A, const ok_float * B,
	const enum CBLAS_ORDER ord)
	{ matrix_memcpy_ma_<ok_float>(A, B, ord); }

void matrix_memcpy_am(ok_float * A, const matrix * B,
	const enum CBLAS_ORDER ord)
	{ matrix_memcpy_am_<ok_float>(A, B, ord); }

void matrix_print(matrix * A)
{
	ok_float row_host[A->size2];
	vector row = (vector){0, 0, OK_NULL};

	for (uint i = 0; i < A->size1; ++i) {
		matrix_row(&row, A, i);
		vector_memcpy_av(row_host, &row, 1);
		for (uint j = 0; j < A->size2; ++j)
			printf("%0.2e ", row_host[j]);
		printf("\n");
	}
	printf("\n");
}

void matrix_scale(matrix * A, ok_float x)
{
	size_t i;
	vector row_col;
	row_col.data = OK_NULL;

	if (A->order == CblasRowMajor)
		for(i = 0; i < A->size1; ++i) {
			matrix_row(&row_col, A, i);
			vector_scale(&row_col, x);
		}
	else
		for(i = 0; i < A->size2; ++i) {
			matrix_column(&row_col, A, i);
			vector_scale(&row_col, x);
		}
}

void matrix_scale_left(matrix * A, const vector * v)
{
	size_t i;
	vector col;
	col.data = OK_NULL;

	for(i = 0; i < A->size2; ++i) {
		matrix_column(&col, A, i);
		vector_mul(&col, v);
	}
}

void matrix_scale_right(matrix * A, const vector * v)
{
	size_t i;
	vector row;
	row.data = OK_NULL;

	for(i = 0; i < A->size1; ++i) {
		matrix_row(&row, A, i);
		vector_mul(&row, v);
	}
}

void matrix_abs(matrix * A)
{
	size_t i;
	vector row_col = (vector){0,0,OK_NULL};
	if (A->order == CblasRowMajor)
		for(i = 0; i < A->size1; ++i) {
			matrix_row(&row_col, A, i);
			vector_abs(&row_col);
		}
	else
		for(i = 0; i < A->size2; ++i) {
			matrix_column(&row_col, A, i);
			vector_abs(&row_col);
		}
}

void matrix_pow(matrix * A, const ok_float x)
{
	size_t i;
	vector row_col = (vector){0,0,OK_NULL};
	if (A->order == CblasRowMajor)
		for(i = 0; i < A->size1; ++i) {
			matrix_row(&row_col, A, i);
			vector_pow(&row_col, x);
		}
	else
		for(i = 0; i < A->size2; ++i) {
			matrix_column(&row_col, A, i);
			vector_pow(&row_col, x);
		}
}

#ifdef __cplusplus
}
#endif