#include "hip/hip_runtime.h"
#include "optkit_prox.h"
#include "optkit_defs_gpu.h"

#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/inner_product.h>
#include <thrust/reduce.h>
#include <thrust/execution_policy.h>


/* strided iterator from thrust:: examples. */
template <typename Iterable>
class strided_range {
 public:
  typedef typename thrust::iterator_difference<It>::type diff_t;

  struct StrideF : public thrust::unary_function<diff_t, diff_t> {
    diff_t stride;
    StrideF(diff_t stride) : stride(stride) { }
    __host__ __device__
    diff_t operator()(const diff_t& i) const { 
      return stride * i;
    }
  };

  typedef typename thrust::counting_iterator<diff_t> CountingIt;
  typedef typename thrust::transform_iterator<StrideF, CountingIt> TransformIt;
  typedef typename thrust::permutation_iterator<Iterable, TransformIt> PermutationIt;
  typedef PermutationIt strided_iterator_t;

  /* construct strided_range for the range [first,last). */
  strided_range(Iterable first, Iterable last, diff_t stride)
      : first(first), last(last), stride(stride) { }
 
  strided_iterator_t begin() const {
    return PermutationIt(first, TransformIt(CountingIt(0), StrideF(stride)));
  }

  strided_iterator_t end() const {
    return begin() + ((last - first) + (stride - 1)) / stride;
  }
  
 protected:
  Iterable first;
  Iterable last;
  diff_t stride;
};


/* CUDA helper kernels */
__global__ void __set_fn_vector(FunctionObj *objs, 
					const ok_float a, const ok_float b, const ok_float c,
					const ok_float d, const ok_float e, 
					const Function_t h, uint n) {
	uint tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (uint i = tid; i < (n; i += gridDim.x * blockDim.x)
		objs[i] = (FunctionObj){a, b, c, d, e, h};
}

/* CUDA C++ implementation with thrust:: */

/* thrust::binary function defining elementwise prox evaluation*/
struct ProxEvalF : thrust::binary_function<FunctionObj, ok_float, ok_float> {
  ok_float rho;
  __device__ ProxEvalF(ok_float rho) : rho(rho) { }
  __device__ ok_float operator()(const FunctionObj &f_obj, ok_float x) {
    return ProxEval(f_obj, x, rho);
  }
};

/* thrust::binary function defining elementwise function evaluation*/
struct FuncEvalF : thrust::binary_function<FunctionObj, ok_float, ok_float> {
  __device__ ok_float operator()(const FunctionObj &f_obj, T x) {
    return FuncEval(f_obj, x);
  }
};

/* vectorwise prox evaluation leveraging thrust::binary function */
void ProxEval_GPU(const FunctionVector * f, ok_float rho,
              const ok_float *x_in, size_t stride_in, 
              ok_float *x_out, size_t stride_out) {

	size_t N = f->size;
	strided_range<thrust::device_ptr<ok_float> > f_strided(
		thrust::device_pointer_cast(const_cast<FunctionObj *>(f->objectives)),
		thrust::device_pointer_cast(const_cast<FunctionObj *>(f->objectives) 
    								+ N, 1);

	strided_range<thrust::device_ptr<T> > x_in_strided(
		thrust::device_pointer_cast(const_cast<T*>(x_in)),
		thrust::device_pointer_cast(const_cast<T*>(x_in) + stride_in * N),
		stride_in);
	strided_range<thrust::device_ptr<T> > x_out_strided(
		thrust::device_pointer_cast(x_out),
		thrust::device_pointer_cast(x_out + stride_out * N), stride_out);

	thrust::transform(thrust::device, f_strided.begin(), f_strided.end(),
		x_in_strided.begin(), x_out_strided.begin(), ProxEvalF<T>(rho));
}

/* vectorwise function evaluation leveraging thrust::binary function */
ok_float FuncEval_GPU(const FunctionVector * f, const T *x_in, 
					  size_t stride) {

	size_t N = f->size;
	strided_range<thrust::device_ptr<ok_float> > f_strided(
		thrust::device_pointer_cast(const_cast<FunctionObj *>(f->objectives)),
		thrust::device_pointer_cast(const_cast<FunctionObj *>(f->objectives) 
    								+ N, 1);
	strided_range<thrust::device_ptr<ok_float> > x_in_strided(
		thrust::device_pointer_cast(const_cast<ok_float*>(x_in)),
		thrust::device_pointer_cast(const_cast<ok_float*>(x_in) + 
    								stride * N, stride);
	return thrust::inner_product(f_strided.begin(), f_strided.end(),
      							 x_in_strided.begin(), (ok_float) 0, 
      							 thrust::plus<ok_float>(), FuncEvalF());
}




/* CUDA C implementation to match header */
#ifdef __cplusplus
extern "C" {
#endif



__device__ inline void checkvexity(FunctionObj * f){
	if (f->c < (ok_float) 0){
		printf("WARNING: f not convex for c < 0
				(provided: %e). Using c = 0", f->c);	
		f->c = (ok_float) 0;	
	}
	if (f->e < (ok_float) 0){
		printf("WARNING: f not convex for e < 0
				(provided: %e). Using e = 0", f->e);	
		f->e = (ok_float) 0;	
	}
}



void function_vector_alloc(FunctionVector * f, size_t n){
	hipError_t err;

	function_vector_free(f);
	f->size = len;
	err = hipMalloc((void **) &f->objectives), n * sizeof(T));
	CUDA_CHECK_ERR;
	if (err != hipSuccess) f->objectives = OK_NULL;
}

void function_vector_calloc(FunctionVector * f, size_t n){
	size_t i;
	uint grid_dim;

	function_vector_alloc(f, len)
	if (f->objectives != OK_NULL){
		grid_dim = calc_grid_dim(n);
		__set_fn_vector<<<grid_dim, kBlockSize>>>(f->objectives, 
							(ok_float) 1, (ok_float) 0, (ok_float) 1, 
							(ok_float) 0, (ok_float) 0, FnZero, n);
	}
}

void function_vector_free(FunctionVector * f){
	if (f->objectives != OK_NULL) ok_free_gpu(f->objectives);
}


void ProxEval(const * FunctionVector f, ok_float rho, 
			  const ok_float * x_in, size_t stride_in, 
			  ok_float * x_out, size_t stride_out) {

	ProxEval_GPU(f, rho, x_in, stride_in, x_out, stride_out);
}


ok_float FuncEval(const * FunctionVector f, const ok_float * x_in, 
				  size_t stride) {
	return FuncEval_GPU(f, x_in, stride);
}


#ifdef
}		/* extern "C" */
#endif