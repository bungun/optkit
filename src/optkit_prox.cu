#include "hip/hip_runtime.h"
#include "optkit_prox.h"
#include "optkit_defs_gpu.h"



/* CUDA helper kernels */
__global__ void 
__set_fn_vector(FunctionObj * objs, 
					const ok_float a, const ok_float b, const ok_float c,
					const ok_float d, const ok_float e, 
					const Function_t h, uint n) {
	uint tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (uint i = tid; i < n; i += gridDim.x * blockDim.x)
		objs[i] = (FunctionObj){
			.a = a, 
			.b = b, 
			.c = c, 
			.d = d, 
			.e = e, 
			.h = h
		};
}

/* CUDA C++ implementation with thrust:: */

/* thrust::binary function defining elementwise prox evaluation*/
struct ProxEvalF : thrust::binary_function<FunctionObj, ok_float, ok_float> {
  ok_float rho;
  __device__ ProxEvalF(ok_float rho) : rho(rho) { }
  __device__ ok_float operator()(const FunctionObj f_obj, ok_float x) {
    return ProxEval(&f_obj, x, rho);
  }
};

/* thrust::binary function defining elementwise function evaluation*/
struct FuncEvalF : thrust::binary_function<FunctionObj *, ok_float, ok_float> {
  __device__ ok_float operator()(const FunctionObj * f_obj, ok_float x) {
    return FuncEval(&f_obj, x);
  }
};

/* vectorwise prox evaluation leveraging thrust::binary function */
void 
ProxEval_GPU(const FunctionVector * f, ok_float rho, 
	const vector * x_in, vector *x_out){
	size_t N = f->size;
	strided_range<thrust::device_ptr<FunctionObj> > f_strided(
		thrust::device_pointer_cast(f->objectives),
		thrust::device_pointer_cast(f->objectives + N), 1);

	strided_range<thrust::device_ptr<ok_float> > x_in_strided(
		thrust::device_pointer_cast(x_in->data),
		thrust::device_pointer_cast(x_in->data + x_in->stride * N), 
		x_in->stride);
	strided_range<thrust::device_ptr<ok_float> > x_out_strided(
		thrust::device_pointer_cast(x_out->data),
		thrust::device_pointer_cast(x_out->data + x_out->stride * N),
		x_out->stride);

	thrust::transform(thrust::device, f_strided.begin(), f_strided.end(),
		x_in_strided.begin(), x_out_strided.begin(), ProxEvalF(rho));
}

/* vectorwise function evaluation using thrust::binary_function */
ok_float 
FuncEval_GPU(const FunctionVector * f, const vector * x) {

	size_t N = f->size;
	strided_range<thrust::device_ptr<FunctionObj> > f_strided(
		thrust::device_pointer_cast(f->objectives),
		thrust::device_pointer_cast(f->objectives + N), 1);
	strided_range<thrust::device_ptr<ok_float> > x_strided(
		thrust::device_pointer_cast(x->data),
		thrust::device_pointer_cast(x->data + x->stride * N), x->stride);
	return thrust::inner_product(f_strided.begin(), f_strided.end(),
      							 x_strided.begin(), (ok_float) 0, 
      							 thrust::plus<ok_float>(), FuncEvalF());
}


/* CUDA C implementation to match header */
#ifdef __cplusplus
extern "C" {
#endif



__device__ inline void 
checkvexity(FunctionObj * f){
	if (f->c < (ok_float) 0){
		printf("WARNING: f not convex for c < 0 (provided: %e). Using c = 0" \
			, f->c);	
		f->c = (ok_float) 0;	
	}
	if (f->e < (ok_float) 0){
		printf("WARNING: f not convex for e < 0 (provided: %e). Using e = 0" \
			, f->e);	
		f->e = (ok_float) 0;	
	}
}


void 
function_vector_alloc(FunctionVector * f, size_t n){
	function_vector_free(f);
	f->size = n;
	ok_alloc_gpu(f->objectives, n * sizeof(ok_float));
	CUDA_CHECK_ERR;
	//if (err != hipSuccess) f->objectives = OK_NULL;
}

void 
function_vector_calloc(FunctionVector * f, size_t n){
	uint grid_dim;

	function_vector_alloc(f, n);
	if (f->objectives != OK_NULL){
		grid_dim = calc_grid_dim(n);
		__set_fn_vector<<<grid_dim, kBlockSize>>>(f->objectives, 
							(ok_float) 1, (ok_float) 0, (ok_float) 1, 
							(ok_float) 0, (ok_float) 0, FnZero, n);
	}
}

void 
function_vector_free(FunctionVector * f){
	if (f->objectives != OK_NULL) ok_free_gpu(f->objectives);
}


void 
ProxEvalVector(const FunctionVector * f, ok_float rho,
			  const vector * x_in, vector * x_out){
	ProxEval_GPU(f, rho, x_in, x_out);
}

ok_float 
FuncEvalVector(const FunctionVector * f, const vector * x){
	return FuncEval_GPU(f, x);
}


#ifdef __cplusplus
}		/* extern "C" */
#endif